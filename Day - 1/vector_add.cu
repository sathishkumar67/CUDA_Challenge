#include <iostream>
#include <hip/hip_runtime.h>

#define N 128  // Vector dimension

__global__ void vectorAdd(const float *A, const float *B, float *C) {
    int idx = threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    float h_A[N], h_B[N], h_C[N]; // Host vectors
    float *d_A, *d_B, *d_C; // Device vectors

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(N - i);
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with one block of N threads
    vectorAdd<<<1, N>>>(d_A, d_B, d_C);

    // Copy result back to host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print some results
    std::cout << "Result: ";
    for (int i = 0; i < N; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}