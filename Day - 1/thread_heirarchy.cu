
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printThreadInfo() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Thread ID: %d, Block ID: %d, Block Dim: %d\n", idx, blockIdx.x, blockDim.x);
    __syncthreads();
}


int main() {
    int blocks, threadsPerBlock;
    blocks = 2;
    threadsPerBlock = 4;
    printThreadInfo<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    return 0;
}