
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU! Thread: %d, Block: %d\n", threadIdx.x, blockIdx.x);
}

int main() {
    helloFromGPU<<<2, 5>>>();  // Launch kernel with 2 blocks and 5 threads each
    hipDeviceSynchronize();  // Ensure all GPU prints complete
    return 0;
}
